#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "ClusterAssign.h"

__global__ void calculateDistancesKernel(
    const double* pointsX, const double* pointsY, int numPoints,
    const double* centroidsX, const double* centroidsY, int numClusters,
    int* clusterAssignments) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numPoints) {
        double x = pointsX[idx];
        double y = pointsY[idx];
        double minDist = (x - centroidsX[0]) * (x - centroidsX[0]) +
                         (y - centroidsY[0]) * (y - centroidsY[0]);
        int bestCluster = 0;

        for (int j = 1; j < numClusters; ++j) {
            double dist = (x - centroidsX[j]) * (x - centroidsX[j]) +
                          (y - centroidsY[j]) * (y - centroidsY[j]);
            if (dist < minDist) {
                minDist = dist;
                bestCluster = j;
            }
        }
        clusterAssignments[idx] = bestCluster;
        printf("Thread %d: Point (%f, %f), assigned to cluster %d\n", idx, pointsX[idx], pointsY[idx], bestCluster);
        
    }
    
    if (idx == 0) {
    printf("Kernel running for %d points and %d clusters\n", numPoints, numClusters);
    }

}

extern "C" void calculateDistancesCUDA(
    const double* hostPointsX, const double* hostPointsY, int numPoints,
    const double* hostCentroidsX, const double* hostCentroidsY, int numClusters,
    int* hostClusterAssignments) {
    
    // Device memory allocation
    double *d_pointsX, *d_pointsY, *d_centroidsX, *d_centroidsY;
    int* d_clusterAssignments;

    hipError_t err;

    err = hipMalloc(&d_pointsX, numPoints * sizeof(double));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for pointsX: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc(&d_pointsY, numPoints * sizeof(double));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for pointsY: %s\n", hipGetErrorString(err));
        hipFree(d_pointsX);
        return;
    }

    err = hipMalloc(&d_centroidsX, numClusters * sizeof(double));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for centroidsX: %s\n", hipGetErrorString(err));
        hipFree(d_pointsX);
        hipFree(d_pointsY);
        return;
    }

    err = hipMalloc(&d_centroidsY, numClusters * sizeof(double));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for centroidsY: %s\n", hipGetErrorString(err));
        hipFree(d_pointsX);
        hipFree(d_pointsY);
        hipFree(d_centroidsX);
        return;
    }

    err = hipMalloc(&d_clusterAssignments, numPoints * sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for clusterAssignments: %s\n", hipGetErrorString(err));
        hipFree(d_pointsX);
        hipFree(d_pointsY);
        hipFree(d_centroidsX);
        hipFree(d_centroidsY);
        return;
    }

    // Copy data to device
    hipMemcpy(d_pointsX, hostPointsX, numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_pointsY, hostPointsY, numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidsX, hostCentroidsX, numClusters * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroidsY, hostCentroidsY, numClusters * sizeof(double), hipMemcpyHostToDevice);

    // Kernel launch
    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;
    //printf("Launching kernel with %d blocks and %d threads per block\n", numBlocks, blockSize);

    calculateDistancesKernel<<<numBlocks, blockSize>>>(
        d_pointsX, d_pointsY, numPoints, d_centroidsX, d_centroidsY, numClusters, d_clusterAssignments);

    // Check for errors during kernel launch
hipError_t err2 = hipGetLastError();
if (err2 != hipSuccess) {
    printf("Error launching kernel: %s\n", hipGetErrorString(err2));
}
    
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA kernel execution failed: %s\n", hipGetErrorString(err));
    } else {
    printf("Kernel executed successfully.\n");
}

    // Copy results back to host
    hipMemcpy(hostClusterAssignments, d_clusterAssignments, numPoints * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_pointsX);
    hipFree(d_pointsY);
    hipFree(d_centroidsX);
    hipFree(d_centroidsY);
    hipFree(d_clusterAssignments);
}

